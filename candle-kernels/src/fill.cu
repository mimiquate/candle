
#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"
#include<stdint.h>

template<typename T>
__device__ void fill_with(T *buf, T value, const size_t numel) {
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
        buf[i] = value;
    }
}
extern "C" __global__ void fill_u8(uint8_t *buf, uint8_t value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_f16(__half *buf, __half value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_f32(float *buf, float value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_f64(double *buf, double value, const size_t numel) { fill_with(buf, value, numel); }
